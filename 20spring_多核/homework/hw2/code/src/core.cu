#include "hip/hip_runtime.h"
/*###################################################
##  文件说明：
##  CUDA相关函数，kernal函数，callback函数的实现
#####################################################*/

#include "core.h"

/*#####################################
## info类
## 记录当前最短距离信息
## dis：最短距离
## pos：最短距离对应点的编号
#####################################*/
struct info{
    float dis;
    int pos; 
};

/*#####################################
## kernal
## CUDA的kernal函数的实现，实现了最终版
## 的kernal，支持NMK三个维度的优化，同时
## 也添加了共享内存优化。
## params:
## n: 参考坐标数
## m: 询问坐标数
## D: 空间维度
## PartNum: m维的分割数
## DimNum: D维的分割数
## qry_d: 询问坐标
## refer_d: 参考坐标
## output_d: 输出结果
#####################################*/
__global__ void kernal(int n, int m, int D, int PartNum, int DimNum, float *qry_d, float *refer_d, int *output_d) {
    int idx = blockIdx.x;
    int ThreadId = threadIdx.x;
    int part = ThreadId / DimNum;
    int dim = ThreadId % DimNum;
    int BlkSize = (n+PartNum-1)/PartNum;
    int DimSize = (D+DimNum-1)/DimNum;
    int L = BlkSize*part, R = min(L+BlkSize, n);
    int _l = DimSize*dim, _r = min(_l+DimSize, D);
    
    extern __shared__ info Str[];
    info *Info = Str;
    float *sum = (float*)&Info[PartNum];
    float *Cur = (float*)&sum[PartNum*DimNum];
    
    if(part==0){
        for(int i=_l;i<_r;i++) Cur[i] = qry_d[idx*D+i];
    }
    __syncthreads();
    
    float ans = -1;
    int pos = -1;
    for(int i=L;i<R;i++)
    {
        float dis = 0, diff;
        for(int j=_l;j<_r;j++) {
            diff = Cur[j] - refer_d[i*D+j];
            dis += diff * diff;
        }
        
        sum[ThreadId] = dis;
        __syncthreads();
        
        if(!dim){
            for(int j=1;j<DimNum;j++) dis += sum[ThreadId+j];
            if(ans<0 || dis<ans) ans=dis, pos=i;
        }
    }
    if(!dim) Info[part] = (info){ans, pos};
    __syncthreads();
    
    if(!dim) {
        for(int stride=(PartNum>>1);stride>0;stride>>=1) {
            if(part+stride<PartNum){
                if(Info[part].dis > Info[part+stride].dis)
                    Info[part] = Info[part+stride];
            }
            __syncthreads();
        }
    }
    
    if(!ThreadId) output_d[idx] = Info[0].pos;
}

/*#####################################
## kernal_ns
## CUDA的kernal函数的实现，实现了最终版
## 的kernal，支持NMK三个维度的优化，不
## 添加了共享内存优化。
## params:
## n: 参考坐标数
## m: 询问坐标数
## D: 空间维度
## PartNum: m维的分割数
## DimNum: D维的分割数
## qry_d: 询问坐标
## refer_d: 参考坐标
## output_d: 输出结果
#####################################*/
__global__ void kernal_ns(int n, int m, int D, int PartNum, int DimNum, float *qry_d, float *refer_d, int *output_d) {
    int idx = blockIdx.x;
    int ThreadId = threadIdx.x;
    int part = ThreadId / DimNum;
    int dim = ThreadId % DimNum;
    int BlkSize = (n+PartNum-1)/PartNum;
    int DimSize = (D+DimNum-1)/DimNum;
    int L = BlkSize*part, R = min(L+BlkSize, n);
    int _l = DimSize*dim, _r = min(_l+DimSize, D);
    
    extern __shared__ info Str[];
    info *Info = Str;
    float *sum = (float*)&Info[PartNum];
    
    float ans = -1;
    int pos = -1;
    for(int i=L;i<R;i++)
    {
        float dis = 0, diff;
        for(int j=_l;j<_r;j++) {
            diff = qry_d[idx*D+j] - refer_d[i*D+j];
            dis += diff * diff;
        }
        
        sum[ThreadId] = dis;
        __syncthreads();
        
        if(!dim){
            for(int j=1;j<DimNum;j++) dis += sum[ThreadId+j];
            if(ans<0 || dis<ans) ans=dis, pos=i;
        }
    }
    if(!dim) Info[part] = (info){ans, pos};
    __syncthreads();
    
    if(!dim) {
        for(int stride=(PartNum>>1);stride>0;stride>>=1) {
            if(part+stride<PartNum){
                if(Info[part].dis > Info[part+stride].dis)
                    Info[part] = Info[part+stride];
            }
            __syncthreads();
        }
    }
    
    if(!ThreadId) output_d[idx] = Info[0].pos;
}

/*#####################################
## cudaCallbackCPU
## 串行方法，作为对照，参考样例代码
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
#####################################*/
extern void cudaCallbackCPU(int k, int m, int n, float *searchPoints,
                            float *referencePoints, int **results) {

    int *tmp = (int*)malloc(sizeof(int)*m);
    int minIndex;
    float minSquareSum, diff, squareSum;

    // Iterate over all search points
    for (int mInd = 0; mInd < m; mInd++) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k*mInd+kInd] - referencePoints[k*nInd+kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        tmp[mInd] = minIndex;
    }

    *results = tmp;
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}

/*#####################################
## cudaCallback
## GPU版本callback函数，内存拷贝，调用内核
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
## dim1: M维优化线程数
## dim2: K维优化线程数
## use_share：是否使用内存优化，默认使用
#####################################*/
extern void cudaCallback(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results,
                         int dim1, int dim2, bool use_share = true) {
    float *qry_d, *refer_d;
    int *output_d;
    int size_m = m*k;
    int size_n = n*k;
    
    CHECK(hipMalloc((void **)&qry_d, sizeof(float)*size_m));
    CHECK(hipMalloc((void **)&refer_d, sizeof(float)*size_n));
    CHECK(hipMalloc((void **)&output_d, sizeof(int)*m));
    CHECK(hipMemcpy(qry_d, searchPoints, sizeof(float)*size_m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(refer_d, referencePoints, sizeof(float)*size_n, hipMemcpyHostToDevice));
    
    int BlkNum = m;
    int PartNum = min(dim1, n);
    int DimNum = min(dim2, (k+1)/2);
    int ThreadNum = PartNum * DimNum;
    int SizeInfo = sizeof(info) * PartNum;
    int SizeSum = sizeof(float) * (PartNum * DimNum);
    int SizeCur = sizeof(float) * k;
    if(use_share) {
        kernal<<< BlkNum, ThreadNum, SizeInfo + SizeSum + SizeCur >>>(n, m, k, PartNum, DimNum, qry_d, refer_d, output_d);
    } else {
        kernal_ns<<< BlkNum, ThreadNum, SizeInfo + SizeSum >>>(n, m, k, PartNum, DimNum, qry_d, refer_d, output_d);
    }
    
    *results = (int *)malloc(sizeof(int)*m);
    CHECK(hipMemcpy(*results, output_d, sizeof(int)*m, hipMemcpyDeviceToHost));
    CHECK(hipFree(qry_d));
    CHECK(hipFree(refer_d));
    
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}


/*##########################################
## cudaCallback1
## baseline策略, dim1=1, dim2=1, 不使用内存优化
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
##########################################*/
extern void cudaCallback1(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results) {
    cudaCallback(k, m, n, searchPoints, referencePoints, results, 1, 1, false);
}

/*##########################################
## cudaCallback2
## baseline策略, dim1=1, dim2=1, 使用内存优化
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
##########################################*/
extern void cudaCallback2(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results) {
    cudaCallback(k, m, n, searchPoints, referencePoints, results, 1, 1);
}

/*##########################################
## cudaCallback3
## 在M维优化的策略, dim1=128, dim2=1, 使用内存优化
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
##########################################*/
extern void cudaCallback3(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results) {
    cudaCallback(k, m, n, searchPoints, referencePoints, results, 128, 1);
}

/*##########################################
## cudaCallback4
## 在MK两维分别优化的策略, dim1=128, dim2=4, 不使用内存优化
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
##########################################*/
extern void cudaCallback4(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results) {
    
    cudaCallback(k, m, n, searchPoints, referencePoints, results, 128, 4, false);
}

/*##########################################
## cudaCallback5
## 在MK两维分别优化的策略, dim1=128, dim2=4, 使用内存优化
## params:
## m: 询问坐标数
## n: 参考坐标数
## k: 空间维度
## searchPoints: 询问坐标
## referencePoints: 参考坐标
## results: 输出结果
##########################################*/
extern void cudaCallback5(int k, int m, int n, float *searchPoints,
                         float *referencePoints, int **results) {
    
    cudaCallback(k, m, n, searchPoints, referencePoints, results, 128, 4);
}


