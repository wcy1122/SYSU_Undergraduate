#include "hip/hip_runtime.h"
/*###################################################
##  文件说明：
##  CUDA相关函数，kernal函数，callback函数的实现
#####################################################*/

#include "core.h"
#define N 7000010
__device__ float input_g[N], output_g[N], logs[26];

/*#####################################
## kernel0
## baseline方法的核函数
## params:
## width, height: 矩阵长宽
## input: 输入数据
## output: 运算结果
#####################################*/
__global__ void kernel0(int width, int height, float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx / height, y = idx % height;
    int L = max(0, x-2), R = min(x+3, width);
    int D = max(0, y-2), U = min(y+3, height);
    int all_idxs = (R-L) * (U-D);
    float ans = 0, prob;
    
    int Cnt;
    for(int k=0;k<16;k++) {
        Cnt = 0;
        for(int i=L;i<R;i++)
            for(int j=D;j<U;j++) {
                if(input[i * height + j]==k) Cnt++;
            }
        if(Cnt) {
            prob = 1.0f * Cnt / all_idxs;
            ans -= prob * logf(prob);
        }
    }
    output[idx] = ans;
}

/*#####################################
## kernel1
## MemTime方法的核函数
## params:
## width, height: 矩阵长宽
## input: 输入数据
## output: 运算结果
#####################################*/
__global__ void kernel1(int width, int height, float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx / height, y = idx % height;
    int L = max(0, x-2), R = min(x+3, width);
    int D = max(0, y-2), U = min(y+3, height);
    int all_idxs = (R-L) * (U-D);
    float ans = 0, prob;
    
    char Cnt[16] = {0};
    for(int i=L;i<R;i++)
        for(int j=D;j<U;j++) {
            Cnt[(int)input[i * height + j]]++;
        }
    for(int i=0;i<16;i++)
        if(Cnt[i]) {
            prob = 1.0f * Cnt[i] / all_idxs;
            ans -= prob * logf(prob);
        }
    output[idx] = ans;
}

/*#####################################
## kernel2
## PreLog方法的核函数
## params:
## width, height: 矩阵长宽
## input: 输入数据
## output: 运算结果
#####################################*/
__global__ void kernel2(int width, int height, float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx / height, y = idx % height;
    int L = max(0, x-2), R = min(x+3, width);
    int D = max(0, y-2), U = min(y+3, height);
    int all_idxs = (R-L) * (U-D);
    float ans = 0, prob;
    
    char Cnt[16] = {0};
    for(int i=L;i<R;i++)
        for(int j=D;j<U;j++) {
            Cnt[(int)input[i * height + j]]++;
        }
    float log_base = logs[all_idxs];
    for(int i=0;i<16;i++)
        if(Cnt[i]) {
            prob = 1.0f * Cnt[i] / all_idxs;
            ans -= prob * (logs[Cnt[i]] - log_base);
        }
    output[idx] = ans;
}

/*#####################################
## kernel3
## ShareMemo(使用共享内存)方法的核函数
## params:
## width, height: 矩阵长宽
## input: 输入数据
## output: 运算结果
## ThreadNum: 线程数
#####################################*/
__global__ void kernel3(int width, int height, float *input, float *output, int ThreadNum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int p = threadDim.x;
    int x = idx / height, y = idx % height;
    int L = max(0, x-2), R = min(x+3, width);
    int D = max(0, y-2), U = min(y+3, height);
    int all_idxs = (R-L) * (U-D);
    float ans = 0, prob;
    
    extern __shared__ float Cur[6500];
    int sid = threadIdx.x;
    
    char Cnt[16] = {0};
    for(int i=L;i<R;i++){
        Cur[sid] = input[i * height + y];
        __syncthreads();
        if(y<2 || y>=height-2 || sid<2 || sid+2>=ThreadNum) {
            for(int j=D;j<U;j++) {
                Cnt[(int)input[i * height + j]]++;
            }
        }
        else {
            for(int j=-2;j<=2;j++) {
                Cnt[(int)Cur[sid + j]]++;
            }
        }
    }
    float log_base = logs[all_idxs];
    for(int i=0;i<16;i++)
        if(Cnt[i]) {
            prob = 1.0f * Cnt[i] / all_idxs;
            ans -= prob * (logs[Cnt[i]] - log_base);
        }
    output[idx] = ans;
}

/*#####################################
## kernel4
## StaticMemo(使用全局内存)方法的核函数
## params:
## width, height: 矩阵长宽
#####################################*/
__global__ void kernel4(int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx / height, y = idx % height;
    int L = max(0, x-2), R = min(x+3, width);
    int D = max(0, y-2), U = min(y+3, height);
    int all_idxs = (R-L) * (U-D);
    float ans = 0, prob;
    
    char Cnt[16] = {0};
    for(int i=L;i<R;i++)
        for(int j=D;j<U;j++) {
            Cnt[(int)input_g[i * height + j]]++;
        }
    float log_base = logs[all_idxs];
    for(int i=0;i<16;i++)
        if(Cnt[i]) {
            prob = 1.0f * Cnt[i] / all_idxs;
            ans -= prob * (logs[Cnt[i]] - log_base);
        }
    output_g[idx] = ans;
}

/*#####################################
## prepare_log
## 预处理log
#####################################*/
void prepare_log() {
    float _logs[26];
    for(int i=1;i<=25;i++) _logs[i] = logf(i);
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(logs), _logs, sizeof(float)*26));
}

/*#####################################
## cudaCallback
## CUDA的callback函数，负责数据传输，kernalxuanze
## params:
## width, height: 矩阵长宽
## Flag：选择的方法
## sample: 输入数据
## result: 运算结果
#####################################*/
void cudaCallback(int width, int height, int Flag, float *sample, float **result) {
    int size = width * height;
    float *input_d, *output_d;
    
    if(Flag<4) {
        CHECK(hipMalloc((void **)&input_d, sizeof(float)*size));
        CHECK(hipMalloc((void **)&output_d, sizeof(float)*size));
        CHECK(hipMemcpy(input_d, sample, sizeof(float)*size, hipMemcpyHostToDevice));
    }
    else {
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(input_g), sample, sizeof(float)*size));
    }
    // Allocate device memory and copy data from host to device
    
    // Invoke the device function
    int ThreadNum = divup(size, 1024);
    if(Flag==0){
        kernel0<<< ThreadNum, 1024 >>>(width, height, input_d, output_d);
    } else if(Flag==1) {
        kernel1<<< ThreadNum, 1024 >>>(width, height, input_d, output_d);
    } else if(Flag==2) {
        kernel2<<< ThreadNum, 1024 >>>(width, height, input_d, output_d);
    } else if(Flag==3) {
        kernel3<<< ThreadNum, 1024 >>>(width, height, input_d, output_d, ThreadNum);
    } else {
        kernel4<<< ThreadNum, 1024 >>>(width, height);
    }
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *result = (float *)malloc(sizeof(float)*size);
    if(Flag<4) {
        CHECK(hipMemcpy(*result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    } else {
        CHECK(hipMemcpyFromSymbol(*result, HIP_SYMBOL(output_g), sizeof(float)*size));
    }

    // Note that you don't have to free sample and *result by yourself
}
